/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int small_helper(int x, int y){
	if(x < y){
		return x;
	}
	else{
		return y;
	}
}//end helper function

__global__ void strongestNeighborScan_gpu(int * src, int * oldDst, int * newDst, int * oldWeight, int * newWeight, int * madeChanges, int distance, int numEdges) {
	
	int i ;
	int tid = blockIdx.x * blockDim.x + threadIdx.x; //global index of the thread
	int total_threads = blockDim.x * gridDim.x; //total number of threads
	
	for(i=tid; i < numEdges; i += total_threads){

		if(tid >= numEdges){ //terminate if thread ID is larger than array size
			return;
		}
	
		if(src[i] == src[i-distance]){//if they are in same segments
			if(oldWeight[i] == oldWeight[i-distance]){//if adjacent weights are same
				newDst[i] = small_helper(oldDst[i], oldDst[i-distance]);//update newDst
				newWeight[i] = oldWeight[i];//update newWeight
			}
			else{//if adjacent weights are not same
				newWeight[i] = max(oldWeight[i], oldWeight[i-distance]); //update newWeight
				if(newWeight[i] == oldWeight[i]){ //update newDst
					newDst[i] = oldDst[i];
				}else{
					newDst[i] = oldDst[i-distance];
				} 
			}	
		}

		else{//they are not in same segment
			newWeight[i] = oldWeight[i];
			newDst[i] = oldDst[i];
		}

		//check termination
		if(oldDst[i] != newDst[i]){
			*madeChanges = 1;
		}

	}//end for loop

}//end fucntion