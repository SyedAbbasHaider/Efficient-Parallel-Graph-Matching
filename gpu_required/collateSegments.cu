/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Spring 2020                               *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void collateSegments_gpu(int * src, int * scanResult, int * output, int numEdges) {
	
	int i;
	int tid = blockIdx.x * blockDim.x + threadIdx.x; //global index of the thread
	int total_threads = blockDim.x * gridDim.x; //total number of threads
	
	for(i=tid; i < numEdges; i += total_threads){
		if(src[i] != src[i+1]){
			output[src[i]] = scanResult[i];
		}
	}//end for loop

}//end main function
